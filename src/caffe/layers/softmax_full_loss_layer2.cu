#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftmaxFull2ForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* target_data, Dtype* loss) {
  CUDA_KERNEL_LOOP(index, nthreads) {
	loss[index] = -target_data[index] * log(max(prob_data[index], Dtype(FLT_MIN)));
  }
}

template <typename Dtype>
void SoftmaxFullLoss2Layer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* target_data = bottom[1]->gpu_data();
  const int nthreads = prob_.count();
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  SoftmaxFull2ForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, target_data, loss_data);
      
  Dtype loss;
  for (int i = 0; i < outer_num_; i++) {
    caffe_gpu_asum(inner_num_, loss_data + i * inner_num_, &loss);
    if (normalize_) {
      loss /= prob_.count();
    } else {
      loss /= outer_num_;
    }
    top[0]->mutable_cpu_data()[i] = loss;
  }

  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
void SoftmaxFullLoss2Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to target probs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* target_probs = bottom[1]->gpu_data();

	caffe_gpu_sub(prob_.count(), prob_data, target_probs, bottom_diff);

    for (int i = 0; i < outer_num_; i++) {
      const Dtype loss_weight = top[0]->cpu_diff()[i];
	  Dtype mult = 0;
      if (normalize_) {
	    mult = loss_weight / prob_.count();
      } else {
	    mult = loss_weight / outer_num_;
      }
      caffe_gpu_scal(inner_num_, mult, bottom_diff + i * inner_num_);
	}
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxFullLoss2Layer);

}  // namespace caffe
